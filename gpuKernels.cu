
#include <hip/hip_runtime.h>
//
// Created by François Laenen on 8/07/16.
//

template<typename OP>
__global__ void testArrayWrapper(OP expr){
    printf("from device %g\n", expr[1]);
}


template<typename T, typename OP>
__global__ void kern(T *output, OP expr, const size_t size){
    /// OP2  an expression template
    /// *output a pointer to write to (must be valid, writable from the device)
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size){
        output[i] = expr[i];
    }
}
